#include "hip/hip_runtime.h"
//============================================================================
//  Copyright (c) Kitware, Inc.
//  All rights reserved.
//  See LICENSE.txt for details.
//  This software is distributed WITHOUT ANY WARRANTY; without even
//  the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR
//  PURPOSE.  See the above copyright notice for more information.
//
//  Copyright 2014 Sandia Corporation.
//  Copyright 2014 UT-Battelle, LLC.
//  Copyright 2014. Los Alamos National Security
//
//  Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
//  the U.S. Government retains certain rights in this software.
//
//  Under the terms of Contract DE-AC52-06NA25396 with Los Alamos National
//  Laboratory (LANL), the U.S. Government retains certain rights in
//  this software.
//============================================================================

#define DAX_DEVICE_ADAPTER DAX_DEVICE_ADAPTER_ERROR
#define BOOST_SP_DISABLE_THREADS

// Tests math functions that rely on system math functions in the Cuda runtime
// environment. Ensures that the Cuda versions of the functions are behaving
// the same as the standard C math library functions.

#include <vtkm/cont/cuda/DeviceAdapterCuda.h>

#include <vtkm/math/Compare.h>
#include <vtkm/math/Exp.h>
#include <vtkm/math/Precision.h>
#include <vtkm/math/Sign.h>
#include <vtkm/math/Trig.h>

#include <vtkm/exec/internal/ErrorMessageBuffer.h>

#include <vtkm/exec/Assert.h>

#include <vtkm/cont/cuda/internal/testing/Testing.h>

namespace ut_CudaMath {

#define MY_ASSERT(condition, message) \
  if (!(condition)) \
    { \
    return \
        __FILE__ ":" VTKM_ASSERT_EXEC_STRINGIFY(__LINE__) ": " message \
        " (" #condition ")"; \
    }

template<class Derived>
struct MathTestFunctor
{
  // The original implementation of these kernels just had the tests in the
  // paren operater as you would expect. However, when I modified the test
  // to work in both the control (host) and execution (device) environments,
  // the two had incompatible error reporting mechanisms.  To get arround this
  // problem, I use the paren overload in a curiously recurring template
  // pattern to call the execution-only raise error method in an execution-only
  // method and macros to throw exceptions only in the control environment.

  VTKM_EXEC_EXPORT
  void operator()(vtkm::Id) const
  {
    // Hopefully the derived class will always return constant strings that do
    // not go out of scope. If we get back garbled error strings, this is
    // probably where it happens.
    const char *message = static_cast<const Derived*>(this)->Run();
    if (message != NULL)
      {
      this->ErrorMessage.RaiseError(message);
      }
  }

  vtkm::exec::internal::ErrorMessageBuffer ErrorMessage;
  VTKM_CONT_EXPORT
  void SetErrorMessageBuffer(
      const vtkm::exec::internal::ErrorMessageBuffer &errorMessage)
  {
    this->ErrorMessage = errorMessage;
  }
};

struct TestCompareKernel : public MathTestFunctor<TestCompareKernel>
{
  VTKM_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(vtkm::math::Min(3, 8) == 3, "Got wrong min.");
    MY_ASSERT(vtkm::math::Min(-0.1f, -0.7f) == -0.7f, "Got wrong min.");
    MY_ASSERT(vtkm::math::Max(3, 8) == 8, "Got wrong max.");
    MY_ASSERT(vtkm::math::Max(-0.1f, -0.7f) == -0.1f, "Got wrong max.");
    return NULL;
  }
};

struct TestExpKernel : public MathTestFunctor<TestExpKernel>
{
  VTKM_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(test_equal(vtkm::math::Pow(0.25, 2.0), vtkm::Scalar(0.0625)),
              "Bad power result.");
    MY_ASSERT(test_equal(vtkm::math::Sqrt(3.75),
                         vtkm::math::Pow(3.75, 0.5)),
              "Bad sqrt result.");
    MY_ASSERT(test_equal(vtkm::math::RSqrt(3.75),
                         vtkm::math::Pow(3.75, -0.5)),
              "Bad reciprocal sqrt result.");
    MY_ASSERT(test_equal(vtkm::math::Cbrt(3.75),
                         vtkm::math::Pow(3.75, 1.0/3.0)),
              "Bad cbrt result.");
    MY_ASSERT(test_equal(vtkm::math::RCbrt(3.75),
                         vtkm::math::Pow(3.75, -1.0/3.0)),
              "Bad reciprocal cbrt result.");
    MY_ASSERT(test_equal(vtkm::math::Exp(3.75),
                         vtkm::math::Pow(2.71828183, 3.75)),
              "Bad exp result.");
    MY_ASSERT(test_equal(vtkm::math::Exp2(3.75),
                         vtkm::math::Pow(2.0, 3.75)),
              "Bad exp2 result.");
    MY_ASSERT(test_equal(vtkm::math::ExpM1(3.75),
                         vtkm::math::Pow(2.71828183, 3.75)-vtkm::Scalar(1)),
              "Bad expm1 result.");
    MY_ASSERT(test_equal(vtkm::math::Exp10(3.75),
                         vtkm::math::Pow(10.0, 3.75)),
              "Bad exp2 result.");
    MY_ASSERT(test_equal(vtkm::math::Log2(vtkm::Scalar(0.25)),
                         vtkm::Scalar(-2.0)),
              "Bad value from Log2");
    MY_ASSERT(
          test_equal(vtkm::math::Log2(vtkm::make_Vector4(0.5, 1.0, 2.0, 4.0)),
                     vtkm::make_Vector4(-1.0, 0.0, 1.0, 2.0)),
          "Bad value from Log2");
    MY_ASSERT(test_equal(vtkm::math::Log(vtkm::Scalar(3.75)),
                         vtkm::Scalar(1.321755839982319)),
              "Bad log result.");
    MY_ASSERT(test_equal(vtkm::math::Log10(vtkm::Scalar(3.75)),
                         vtkm::Scalar(0.574031267727719)),
              "Bad log10 result.");
    MY_ASSERT(test_equal(vtkm::math::Log1P(3.75),
                         vtkm::math::Log(4.75)),
              "Bad log1p result.");
    return NULL;
  }
};

struct TestPrecisionKernel : public MathTestFunctor<TestPrecisionKernel>
{
  VTKM_EXEC_CONT_EXPORT const char *Run() const
  {
    vtkm::Scalar zero = 0.0;
    vtkm::Scalar finite = 1.0;
    vtkm::Scalar nan = vtkm::math::Nan();
    vtkm::Scalar inf = vtkm::math::Infinity();
    vtkm::Scalar neginf = vtkm::math::NegativeInfinity();
    vtkm::Scalar epsilon = vtkm::math::Epsilon();

    // General behavior.
    MY_ASSERT(nan != nan, "Nan not equal itself.");
    MY_ASSERT(!(nan >= zero), "Nan not greater or less.");
    MY_ASSERT(!(nan <= zero), "Nan not greater or less.");
    MY_ASSERT(!(nan >= finite), "Nan not greater or less.");
    MY_ASSERT(!(nan <= finite), "Nan not greater or less.");

    MY_ASSERT(neginf < inf, "Infinity big");
    MY_ASSERT(zero < inf, "Infinity big");
    MY_ASSERT(finite < inf, "Infinity big");
    MY_ASSERT(zero > neginf, "-Infinity small");
    MY_ASSERT(finite > neginf, "-Infinity small");

    MY_ASSERT(zero < epsilon, "Negative epsilon");
    MY_ASSERT(finite > epsilon, "Large epsilon");

    // Math check functions.
    MY_ASSERT(!vtkm::math::IsNan(zero), "Bad IsNan check.");
    MY_ASSERT(!vtkm::math::IsNan(finite), "Bad IsNan check.");
    MY_ASSERT(vtkm::math::IsNan(nan), "Bad IsNan check.");
    MY_ASSERT(!vtkm::math::IsNan(inf), "Bad IsNan check.");
    MY_ASSERT(!vtkm::math::IsNan(neginf), "Bad IsNan check.");
    MY_ASSERT(!vtkm::math::IsNan(epsilon), "Bad IsNan check.");

    MY_ASSERT(!vtkm::math::IsInf(zero), "Bad infinity check.");
    MY_ASSERT(!vtkm::math::IsInf(finite), "Bad infinity check.");
    MY_ASSERT(!vtkm::math::IsInf(nan), "Bad infinity check.");
    MY_ASSERT(vtkm::math::IsInf(inf), "Bad infinity check.");
    MY_ASSERT(vtkm::math::IsInf(neginf), "Bad infinity check.");
    MY_ASSERT(!vtkm::math::IsInf(epsilon), "Bad infinity check.");

    MY_ASSERT(vtkm::math::IsFinite(zero), "Bad finite check.");
    MY_ASSERT(vtkm::math::IsFinite(finite), "Bad finite check.");
    MY_ASSERT(!vtkm::math::IsFinite(nan), "Bad finite check.");
    MY_ASSERT(!vtkm::math::IsFinite(inf), "Bad finite check.");
    MY_ASSERT(!vtkm::math::IsFinite(neginf), "Bad finite check.");
    MY_ASSERT(vtkm::math::IsFinite(epsilon), "Bad finite check.");

    MY_ASSERT(test_equal(vtkm::math::FMod(6.5, 2.3), vtkm::Scalar(1.9)),
              "Bad fmod.");
    MY_ASSERT(test_equal(vtkm::math::Remainder(6.5, 2.3),
                         vtkm::Scalar(-0.4)),
              "Bad remainder.");
    vtkm::Scalar remainder, quotient;
    remainder = vtkm::math::RemainderQuotient(6.5, 2.3, quotient);
    MY_ASSERT(test_equal(remainder, vtkm::Scalar(-0.4)), "Bad remainder.");
    MY_ASSERT(test_equal(quotient, vtkm::Scalar(3.0)), "Bad quotient.");
    vtkm::Scalar integral, fractional;
    fractional = vtkm::math::ModF(4.6, integral);
    MY_ASSERT(test_equal(integral, vtkm::Scalar(4.0)), "Bad integral.");
    MY_ASSERT(test_equal(fractional, vtkm::Scalar(0.6)), "Bad fractional.");
    MY_ASSERT(test_equal(vtkm::math::Floor(4.6), vtkm::Scalar(4.0)),
              "Bad floor.");
    MY_ASSERT(test_equal(vtkm::math::Ceil(4.6), vtkm::Scalar(5.0)),
              "Bad ceil.");
    MY_ASSERT(test_equal(vtkm::math::Round(4.6), vtkm::Scalar(5.0)),
              "Bad round.");

    return NULL;
  }
};

struct TestSignKernel : public MathTestFunctor<TestSignKernel>
{
  VTKM_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(vtkm::math::Abs(-1) == 1, "Bad abs.");
    MY_ASSERT(vtkm::math::Abs(vtkm::Scalar(-0.25)) == 0.25, "Bad abs.");
    MY_ASSERT(vtkm::math::IsNegative(-3.1), "Bad negative.");
    MY_ASSERT(!vtkm::math::IsNegative(3.2), "Bad positive.");
    MY_ASSERT(!vtkm::math::IsNegative(0.0), "Bad non-negative.");
    MY_ASSERT(vtkm::math::SignBit(-3.1), "Bad negative SignBit.");
    MY_ASSERT(!vtkm::math::SignBit(3.2), "Bad positive SignBit.");
    MY_ASSERT(!vtkm::math::SignBit(0.0), "Bad non-negative SignBit.");
    MY_ASSERT(vtkm::math::CopySign(-0.25, 100.0) == 0.25, "Copy sign.");

    return NULL;
  }
};

struct TestTrigKernel : public MathTestFunctor<TestTrigKernel>
{
  VTKM_EXEC_CONT_EXPORT const char *Run() const
  {
    MY_ASSERT(test_equal(vtkm::math::Pi(), vtkm::Scalar(3.14159265)),
              "Pi not correct.");

    MY_ASSERT(test_equal(vtkm::math::ATan2(0.0, 1.0),
                         vtkm::Scalar(0.0)),
              "ATan2 x+ axis.");
    MY_ASSERT(test_equal(vtkm::math::ATan2(1.0, 0.0),
                         vtkm::Scalar(0.5*vtkm::math::Pi())),
              "ATan2 y+ axis.");
    MY_ASSERT(test_equal(vtkm::math::ATan2(-1.0, 0.0),
                         vtkm::Scalar(-0.5*vtkm::math::Pi())),
              "ATan2 y- axis.");

    MY_ASSERT(test_equal(vtkm::math::ATan2(1.0, 1.0),
                         vtkm::Scalar(0.25*vtkm::math::Pi())),
              "ATan2 Quadrant 1");
    MY_ASSERT(test_equal(vtkm::math::ATan2(1.0, -1.0),
                         vtkm::Scalar(0.75*vtkm::math::Pi())),
              "ATan2 Quadrant 2");
    MY_ASSERT(test_equal(vtkm::math::ATan2(-1.0, -1.0),
                         vtkm::Scalar(-0.75*vtkm::math::Pi())),
              "ATan2 Quadrant 3");
    MY_ASSERT(test_equal(vtkm::math::ATan2(-1.0, 1.0),
                         vtkm::Scalar(-0.25*vtkm::math::Pi())),
              "ATan2 Quadrant 4");

    vtkm::Scalar angle = (1.0/3.0)*vtkm::math::Pi();
    vtkm::Scalar opposite = vtkm::math::Sqrt(3.0);
    vtkm::Scalar adjacent = 1.0;
    vtkm::Scalar hypotenuse = 2.0;
    MY_ASSERT(test_equal(vtkm::math::Sin(angle), opposite/hypotenuse),
              "Sin failed test.");
    MY_ASSERT(test_equal(vtkm::math::Cos(angle), adjacent/hypotenuse),
              "Cos failed test.");
    MY_ASSERT(test_equal(vtkm::math::Tan(angle), opposite/adjacent),
              "Tan failed test.");
    MY_ASSERT(test_equal(vtkm::math::ASin(opposite/hypotenuse), angle),
              "Arc Sin failed test.");
    MY_ASSERT(test_equal(vtkm::math::ACos(adjacent/hypotenuse), angle),
              "Arc Cos failed test.");
    MY_ASSERT(test_equal(vtkm::math::ATan(opposite/adjacent), angle),
              "Arc Tan failed test.");

    return NULL;
  }
};

template<class Functor>
VTKM_CONT_EXPORT
void TestSchedule(Functor functor)
{
  // Schedule on device.
  vtkm::cont::DeviceAdapterAlgorithm<
      vtkm::cont::DeviceAdapterTagCuda>::Schedule(functor, 1);

  // Run on host. The return value has the same qualification as mentioned
  // before.
  const char *message = functor.Run();
  if (message != NULL)
    {
    DAX_TEST_FAIL(message);
    }
}

VTKM_CONT_EXPORT
void TestCudaMath()
{
  std::cout << "Compare functions" << std::endl;
  TestSchedule(TestCompareKernel());

  std::cout << "Exponential functions" << std::endl;
  TestSchedule(TestExpKernel());

  std::cout << "Precision functions" << std::endl;
  TestSchedule(TestPrecisionKernel());

  std::cout << "Sign functions" << std::endl;
  TestSchedule(TestSignKernel());

  std::cout << "Trig functions" << std::endl;
  TestSchedule(TestTrigKernel());
}

} // namespace ut_CudaMath

//-----------------------------------------------------------------------------
int UnitTestCudaMath(int, char *[])
{
  return vtkm::cont::cuda::internal::Testing::Run(ut_CudaMath::TestCudaMath);
}
